#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <math.h> 

#include <iostream>


#include "stdio.h"

// for debugging
// #define CUDA_ERROR_CHECK
// #define CUDA_TIMING

#define cudaSafeCall( err ) __cnnCudaSafeCall( err, __FILE__, __LINE__ )

inline void __cnnCudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
  if ( hipSuccess != err )
  {
    fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n", file, line, hipGetErrorString( err ) );
    exit( -1 );
  }
#endif
  return;
}


#ifdef CUDA_TIMING
class CudaTimer
{
public:
  CudaTimer() 
  {
    hipEventCreate(&start_);
    hipEventCreate(&stop_);
  }

  ~CudaTimer() 
  {
    hipEventDestroy(start_);
    hipEventDestroy(stop_);
  }

  void start() 
  {
    hipEventRecord(start_, 0);
  }

  float elapsed() 
  {
    hipEventRecord(stop_);
    hipEventSynchronize(stop_);
    float t = 0;
    hipEventElapsedTime(&t, start_, stop_);
    return t;
  }

private:
  hipEvent_t start_;
  hipEvent_t stop_;
};
#endif


// helpers
__forceinline__ __device__ float myabs(const float x)
{
  return fabsf(x);
}

__forceinline__ __device__ double myabs(const double x)
{
  return fabs(x);
}

// CUDA kernels
template <typename T>
__global__ void cuda_primal_update_step_kernel(
  torch::PackedTensorAccessor32<T,3,torch::RestrictPtrTraits> u,
  torch::PackedTensorAccessor32<T,4,torch::RestrictPtrTraits> p,
  torch::PackedTensorAccessor32<T,3,torch::RestrictPtrTraits> ATq,
  const float tau, //torch::PackedTensorAccessor32<T,3,torch::RestrictPtrTraits> tau, //
  const float hz,
  const int Z,
  const int M,
  const int N,
  const int D) 
{
  int ix = blockDim.x * blockIdx.x + threadIdx.x;
  int iy = blockDim.y * blockIdx.y + threadIdx.y;
  int iz = blockDim.z * blockIdx.z + threadIdx.z;

  T temp = 0;
  if (ix < N && iy < M && iz < Z)
  {
    // finite differences backward
    temp += (ix > 0) ? (ix < N - 1) ? p[iz][iy][ix-1][0] - p[iz][iy][ix][0] : p[iz][iy][ix-1][0] : -p[iz][iy][ix][0];
    temp += (iy > 0) ? (iy < M - 1) ? p[iz][iy-1][ix][1] - p[iz][iy][ix][1] : p[iz][iy-1][ix][1] : -p[iz][iy][ix][1]; 
    temp += (iz > 0) ? (iz < Z - 1) ? (p[iz-1][iy][ix][2] - p[iz][iy][ix][2])/hz : p[iz-1][iy][ix][2]/hz : -p[iz][iy][ix][2]/hz;

    u[iz][iy][ix] -= tau*(temp + ATq[iz][iy][ix]);     
    //u[iz][iy][ix] -= tau[iz][iy][ix]*(temp + ATq[iz][iy][ix]);     
  }
}

template <typename T>
__global__ void cuda_dual_update_step_kernel(
  torch::PackedTensorAccessor32<T,4,torch::RestrictPtrTraits> p,
  torch::PackedTensorAccessor32<T,3,torch::RestrictPtrTraits> u,
  torch::PackedTensorAccessor32<T,1,torch::RestrictPtrTraits> sigma,
  const float hz,
  const float lamda,
  const int Z,
  const int M,
  const int N,
  const int D) 
{
  int ix = blockDim.x * blockIdx.x + threadIdx.x;
  int iy = blockDim.y * blockIdx.y + threadIdx.y;
  int iz = blockDim.z * blockIdx.z + threadIdx.z;

  if (ix < N && iy < M && iz < Z)
  {
      const int xp = ix + (ix < N - 1);
      const int yp = iy + (iy < M - 1);
      const int zp = iz + (iz < Z - 1);

      const T ptmp_x = p[iz][iy][ix][0] + sigma[0]*(u[iz][iy][xp] - u[iz][iy][ix]);
      const T ptmp_y = p[iz][iy][ix][1] + sigma[1]*(u[iz][yp][ix] - u[iz][iy][ix]);
      const T ptmp_z = p[iz][iy][ix][2] + sigma[2]*(u[zp][iy][ix] - u[iz][iy][ix])/hz;

      // calculate 2-norm along first dimension
      const T denom_temp = max(sqrtf(ptmp_x*ptmp_x + ptmp_y*ptmp_y + ptmp_z*ptmp_z)/lamda, 1.);

      // prox operation
      p[iz][iy][ix][0] = ptmp_x / denom_temp;
      p[iz][iy][ix][1] = ptmp_y / denom_temp;
      p[iz][iy][ix][2] = ptmp_z / denom_temp;
  }
}

template <typename T>
__global__ void cuda_prox_l2_forward_kernel(
  torch::PackedTensorAccessor32<T,3,torch::RestrictPtrTraits> q,
  torch::PackedTensorAccessor32<T,1,torch::RestrictPtrTraits> sigma,
  const int Z,
  const int M,
  const int N) 
{
  int ix = blockDim.x * blockIdx.x + threadIdx.x;
  int iy = blockDim.y * blockIdx.y + threadIdx.y;
  int iz = blockDim.z * blockIdx.z + threadIdx.z;

  if (ix < N && iy < M && iz < Z)
  {
    // l2 prox on q 
    q[iz][iy][ix] /= (sigma[iz] + 1.);
  }
}

template <typename T>
__global__ void cuda_central_differences_kernel(
  const torch::PackedTensorAccessor32<T,3,torch::RestrictPtrTraits> b,
  const int P,
  const int M,
  const int N,
  torch::PackedTensorAccessor32<T,4,torch::RestrictPtrTraits> Db)
{
  int ix = blockDim.x * blockIdx.x + threadIdx.x;
  int iy = blockDim.y * blockIdx.y + threadIdx.y;
  int iz = blockDim.z * blockIdx.z + threadIdx.z;

  if (ix < N && iy < M && iz < P)
  {
      Db[iz][iy][ix][0] = (ix > 0) ? 
                            (ix < N-1) ? 
                              0.5*(b[iz][iy][ix+1] - b[iz][iy][ix-1]) 
                              : 
                              0.5*(b[iz][iy][ix] - b[iz][iy][ix-1]) 
                            : 
                            0.5*(b[iz][iy][ix+1]-b[iz][iy][ix]);

      Db[iz][iy][ix][1] = (iy > 0) ? 
                            (iy < M-1) ? 
                              0.5*(b[iz][iy+1][ix] - b[iz][iy-1][ix]) 
                              : 
                              0.5*(b[iz][iy][ix] - b[iz][iy-1][ix]) 
                            : 
                            0.5*(b[iz][iy+1][ix]-b[iz][iy][ix]);
  }
  
}


// C++ kernel calls
void cuda_primal_update_step(
  const torch::Tensor &u,
  const torch::Tensor &p,
  const torch::Tensor &ATq,
  const float tau, //const torch::Tensor &tau, //
  const float hz)
{
  TORCH_CHECK(u.dim() == 3, "Expected 3d tensor for dual p");
  TORCH_CHECK(p.dim() == 4, "Expected 4d tensor for primal u");
  TORCH_CHECK(ATq.dim() == 3, "Expected 3d tensor for primal u");
  //TORCH_CHECK(tau.dim() == 3, "Expected 3d tensor for stepsize tau");

  const int Z = p.size(0);
  const int M = p.size(1);
  const int N = p.size(2);
  const int D = p.size(3);
  
  hipDeviceSynchronize();

  const dim3 blockSize(32, 32, 1); 
  const dim3 numBlocks((N + blockSize.x - 1) / blockSize.x,
                      (M + blockSize.y - 1) / blockSize.y,
                      (Z + blockSize.z - 1) / blockSize.z);

#ifdef CUDA_TIMING
  CudaTimer cut;
  cut.start();
#endif

  AT_DISPATCH_FLOATING_TYPES(p.type(), "primal_update_step", ([&]{
    cuda_primal_update_step_kernel<scalar_t><<<numBlocks, blockSize>>>(
      u.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
      p.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
      ATq.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
      tau, //tau.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(), //
      hz,
      Z,
      M,
      N,
      D); 
  }));
  cudaSafeCall(hipGetLastError());

#ifdef CUDA_TIMING
  hipDeviceSynchronize();
  std::cout << "forward time " << cut.elapsed() << std::endl;
#endif
}

void cuda_dual_update_step(
  const torch::Tensor &p,
  const torch::Tensor &u,
  const torch::Tensor &sigma,
  const float hz,
  const float lamda)
{
  TORCH_CHECK(p.dim() == 4, "Expected 4d tensor for dual p");
  TORCH_CHECK(u.dim() == 3, "Expected 3d tensor for primal u");
  TORCH_CHECK(sigma.dim() == 1, "Expected 1d tensor for stepsize sigma");

  const int Z = p.size(0);
  const int M = p.size(1);
  const int N = p.size(2);
  const int D = p.size(3);

  const dim3 blockSize(32, 32, 1); 
  const dim3 numBlocks((N + blockSize.x - 1) / blockSize.x,
                      (M + blockSize.y - 1) / blockSize.y,
                      (Z + blockSize.z - 1) / blockSize.z);

#ifdef CUDA_TIMING
  CudaTimer cut;
  cut.start();
#endif

  AT_DISPATCH_FLOATING_TYPES(p.type(), "dual_update_step", ([&]{
    cuda_dual_update_step_kernel<scalar_t><<<numBlocks, blockSize>>>(
      p.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
      u.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
      sigma.packed_accessor32<scalar_t,1,torch::RestrictPtrTraits>(),
      hz,
      lamda,
      Z,
      M,
      N,
      D); 
  }));
  cudaSafeCall(hipGetLastError());

#ifdef CUDA_TIMING
  hipDeviceSynchronize();
  std::cout << "forward time " << cut.elapsed() << std::endl;
#endif
}

void cuda_prox_l2(
  const torch::Tensor &q,
  const torch::Tensor &sigma)
{
  TORCH_CHECK(q.dim() == 3, "Expected 3d tensor");
  TORCH_CHECK(sigma.dim() == 1, "Expected 1d tensor");

  const int Z = q.size(0);
  const int M = q.size(1);
  const int N = q.size(2);

  const dim3 blockSize(32, 32, 1); 
  const dim3 numBlocks((N + blockSize.x - 1) / blockSize.x,
                      (M + blockSize.y - 1) / blockSize.y,
                      (Z + blockSize.z - 1) / blockSize.z);

#ifdef CUDA_TIMING
  CudaTimer cut;
  cut.start();
#endif

  AT_DISPATCH_FLOATING_TYPES(q.type(), "prox_l2_forward", ([&]{
    cuda_prox_l2_forward_kernel<scalar_t><<<numBlocks, blockSize>>>(
      q.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
      sigma.packed_accessor32<scalar_t,1,torch::RestrictPtrTraits>(),
      Z,
      M,
      N); 
  }));
  cudaSafeCall(hipGetLastError());
  
#ifdef CUDA_TIMING
  hipDeviceSynchronize();
  std::cout << "forward time " << cut.elapsed() << std::endl;
#endif
}

torch::Tensor cuda_nabla2d_cd_forward(
  const torch::Tensor &b)
{
  TORCH_CHECK(b.dim() == 3, "Expected 4d tensor");

  const int P = b.size(0);
  const int M = b.size(1);
  const int N = b.size(2);

  auto Db = torch::zeros({P, M, N, 2}, b.options());

  const dim3 blockSize(32, 32, 1); 
  const dim3 numBlocks((N + blockSize.x - 1) / blockSize.x,
                      (M + blockSize.y - 1) / blockSize.y,
                      (P + blockSize.z - 1) / blockSize.z);

#ifdef CUDA_TIMING
  CudaTimer cut;
  cut.start();
#endif

  AT_DISPATCH_FLOATING_TYPES(b.type(), "nabla2d_cd_forward", ([&]{
    cuda_central_differences_kernel<scalar_t><<<numBlocks, blockSize>>>(
      b.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
      P,
      M,
      N,
      Db.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>());
  }));
  cudaSafeCall(hipGetLastError());

#ifdef CUDA_TIMING
  hipDeviceSynchronize();
  std::cout << "forward time " << cut.elapsed() << std::endl;
#endif

  return Db;
}